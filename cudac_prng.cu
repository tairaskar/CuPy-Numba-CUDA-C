#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#define blockCount 216
#define threadsPerBlock 256
#define sampleCount 1000000000

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

__global__ void setup_kernel(hiprandStateXORWOW *state)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    /* Each thread gets same seed, a different sequence
       number, no offset */
    hiprand_init(1234, id, 0, &state[id]);
}

__global__ void generate_uniform_kernel(hiprandStateXORWOW *state,
                                int n,
                                float *result)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    /* Copy state to local memory for efficiency */
    hiprandStateXORWOW localState = state[id];
    /* Generate pseudo-random uniforms */
    for(int i = id; i < n; i+=threadsPerBlock*blockCount) {
        result[id] = hiprand_uniform(&localState);
    }
}

int main(int argc, char *argv[])
{

    int totalThreads = threadsPerBlock * blockCount;

    hiprandStateXORWOW *devStates;

    float *devResults;


    /* Allocate space for results on device */
    CUDA_CALL(hipMalloc((void **)&devResults, sampleCount *
              sizeof(float)));

    CUDA_CALL(hipMalloc((void **)&devStates, totalThreads *
                  sizeof(hiprandState)));

    setup_kernel<<<blockCount, threadsPerBlock>>>(devStates);


    generate_uniform_kernel<<<blockCount, threadsPerBlock>>>(devStates, sampleCount, devResults);


    CUDA_CALL(hipFree(devStates));

    CUDA_CALL(hipFree(devResults));

    
    return EXIT_SUCCESS;
}

